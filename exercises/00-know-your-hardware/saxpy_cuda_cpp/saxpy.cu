#include "hip/hip_runtime.h"
/**
 * Compute saxpy
 * - on CPU : serial and OpenMP version
 * - on GPU : first using CUDA, then library CuBLAS
 *
 * compare timings.
 *
 */

// =========================
// standard imports
// =========================
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <type_traits> // for std::is_same_v

// =========================
// CUDA imports
// =========================
#include <hip/hip_runtime.h>
#include <hipblas.h>

// =========================
// OpenMP imports
// =========================
#ifdef _OPENMP
#  include <omp.h>
#endif

// =========================
// our imports
// =========================
#include "my_cuda_utils.h"
#include "HostTimer.h"
#include "CudaTimer.h"

// =========================
// global variables and configuration section
// =========================

// number of repetitions of the timing loop
// (CPU timers only have a ~ms resolution)
static int numTimingReps = 100;

#ifdef USE_FLOAT
using real_t = float;
#else
using real_t = double;
#endif

/**
 * @brief Convert a size into a human readable string
 *
 * see
 * https://github.com/NVIDIA/cccl/blob/main/cudax/include/cuda/experimental/__stf/utility/pretty_print.cuh
 */
inline ::std::string
pretty_print_prefix(size_t num_elem)
{
  const char * units[] = { " ", "K", "M", "G", "T" };
  size_t       size = sizeof(units) / sizeof(char *);
  int          i = 0;

  double pretty_size = static_cast<double>(num_elem);
  while (pretty_size >= 1024.0 && static_cast<size_t>(i) < size - 1)
  {
    pretty_size /= 1024.0;
    ++i;
  }

  ::std::ostringstream out;
  out << ::std::fixed << ::std::setprecision(2) << pretty_size << ' ' << units[i];
  return out.str();
}

// =========================
// kernel function (CPU) - serial
// =========================
void
saxpy_serial(int n, real_t alpha, const real_t * x, real_t * y)
{

  for (size_t i = 0; i < n; ++i)
    y[i] = alpha * x[i] + y[i];
}

// =========================
// kernel function (CPU) - OpenMP
// =========================
void
saxpy_openmp(int n, real_t alpha, const real_t * x, real_t * y)
{

#pragma omp parallel for
#pragma ivdep
  for (size_t i = 0; i < n; ++i)
    y[i] = alpha * x[i] + y[i];
}


// =========================
// kernel function (CUDA device)
// =========================
__global__ void
saxpy_cuda(int n, real_t alpha, const real_t * x, real_t * y)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  // except for special cases, the total number of threads in all blocks
  // adds up to more than the vector length n, so this conditional is
  // EXTREMELY important to avoid writing past the allocated memory for
  // the vector y.
  if (i < n)
    y[i] = alpha * x[i] + y[i];
}


// =========================
// main routine
// =========================
int
main(int argc, char ** argv)
{

  // base-2 log of array size
  int log2N = argc > 1 ? atoi(argv[1]) : 22;

  // problem size (vector length) N
  // size_t N = 1234567;
  size_t N = 1 << log2N;
  // size_t N = 40000;

  HostTimer cpuTimer;
  HostTimer ompTimer;
  CudaTimer gpuTimer;

  if constexpr (std::is_same_v<real_t, float>)
    printf("Using data type: float\n");
  else
    printf("Using data type: double\n");

#ifdef _OPENMP
  int         threadId;
#  pragma omp parallel private(threadId)
  {
    threadId = omp_get_thread_num();
    if (threadId == 0)
      printf("Using %d OpenMP threads\n", omp_get_num_threads());
  }
#else
  printf("OpenMP not activated\n");
#endif

  // =========================
  // (1) initialisations:
  //     implemented in my_cuda_utils.c
  // =========================
  initCuda(0);


  // =========================
  // (2) allocate memory on host (main CPU memory) and device,
  //     h_ denotes data residing on the host, d_ on device
  // =========================
  real_t * h_x = (real_t *)malloc(N * sizeof(real_t));
  real_t * h_y = (real_t *)malloc(N * sizeof(real_t));
  real_t * d_x;
  hipMalloc((void **)&d_x, N * sizeof(real_t));
  real_t * d_y;
  hipMalloc((void **)&d_y, N * sizeof(real_t));
  checkErrors("memory allocation");


  // =========================
  // (3) initialise data on the CPU
  // =========================
#pragma omp parallel for
  for (size_t i = 0; i < N; ++i)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (real_t)(N - i + 1);
  }


  // =========================
  // (4) copy data to device
  // =========================
  hipMemcpy(d_x, h_x, N * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, N * sizeof(real_t), hipMemcpyHostToDevice);
  checkErrors("copy data to device");

  real_t alpha = 2.0;

  // =========================
  // (5a) perform computation on host - SERIAL
  //     use our straight forward code
  //     and our utility functions to time everything,
  //     note that gettimeofday has ~ms resolution, so
  //     perform everything in a loop to minimise
  //     timing noise
  // =========================
  cpuTimer.start();
  for (int iter = 0; iter < numTimingReps; ++iter)
    saxpy_serial(N, alpha, h_x, h_y);
  cpuTimer.stop();
  double elapsed = cpuTimer.elapsed();
  printf("CPU CODE (Serial): 2^%d = %8ld (= %s) elements, %10.6f ms per iteration, %6.3f GFLOP/s, "
         "%7.3f GB/s\n",
         log2N,
         N,
         pretty_print_prefix(N).c_str(),
         (elapsed * 1000.0) / (double)numTimingReps,
         2.0 * N * numTimingReps / (elapsed * 1e9),
         3.0 * N * sizeof(real_t) * numTimingReps / (elapsed * 1e9));

  // =========================
  // (5b) perform computation on host - OpenMP
  // =========================
  ompTimer.start();
  for (int iter = 0; iter < numTimingReps; ++iter)
    saxpy_openmp(N, alpha, h_x, h_y);
  ompTimer.stop();
  elapsed = ompTimer.elapsed();
  printf("CPU CODE (OpenMP): 2^%d = %8ld (= %s) elements, %10.6f ms per iteration, %6.3f GFLOP/s, "
         "%7.3f GB/s\n",
         log2N,
         N,
         pretty_print_prefix(N).c_str(),
         (elapsed * 1000.0) / (double)numTimingReps,
         2.0 * N * numTimingReps / (elapsed * 1e9),
         3.0 * N * sizeof(real_t) * numTimingReps / (elapsed * 1e9));


  // =========================
  // (7) perform computation on device, our implementation
  //     use CUDA events to time the execution:
  //     (a) insert "tag" into instruction stream
  //     (b) execute kernel
  //     (c) insert another tag into instruction stream
  //     (d) synchronize (ie, wait for) this tag (event)
  //     CUDA events have a resolution of ~0.5us
  // =========================
  real_t time;

  // Mapping onto the device:
  // - each thread computes one element of the output array in situ
  // - all threads and blocks are independent
  // - use 256 threads per block
  // - use as many blocks as necessary (the last block is not entirely
  //   full if n is not a multiple of 256)
  int numThreadsPerBlock = 128;
  int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

  gpuTimer.start();
  saxpy_cuda<<<numBlocks, numThreadsPerBlock>>>(N, alpha, d_x, d_y);
  gpuTimer.stop();
  time = gpuTimer.elapsed();
  printf("GPU CODE (CUDA)  : 2^%d = %8ld (= %s) elements, %10.6f ms per iteration, %6.3f GFLOP/s, "
         "%7.3f GB/s\n",
         log2N,
         N,
         pretty_print_prefix(N).c_str(),
         time * 1000,
         2.0 * N / (time * 1e9),
         3.0 * N * sizeof(real_t) / (time * 1e9));


  // =========================
  // (8) read back result from device into temp vector
  // =========================
  real_t * h_z = (real_t *)malloc(N * sizeof(real_t));
  hipMemcpy(h_z, d_y, N * sizeof(real_t), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");


  // =========================
  // (9) perform computation on device, CUBLAS
  // =========================
  {
    [[maybe_unused]] hipblasStatus_t status;
    hipblasHandle_t                  handle;
    status = hipblasCreate(&handle);

    gpuTimer.reset();
    if constexpr (std::is_same_v<real_t, float>)
    {
      // printf("%d ", std::is_same<float, real_t>::value);
      gpuTimer.start();
      hipblasSaxpy(handle, N, (float *)&alpha, (float *)d_x, 1, (float *)d_y, 1);
      gpuTimer.stop();
    }
    else if constexpr (std::is_same_v<real_t, double>)
    {
      gpuTimer.start();
      hipblasDaxpy(handle, N, (double *)&alpha, (double *)d_x, 1, (double *)d_y, 1);
      gpuTimer.stop();
    }
    time = gpuTimer.elapsed();
    printf(
      "GPU CODE (CUBLAS): 2^%d = %8ld (= %s) elements, %10.6f ms per iteration, %6.3f GFLOP/s, "
      "%7.3f "
      "GB/s\n",
      log2N,
      N,
      pretty_print_prefix(N).c_str(),
      time * 1000,
      2.0 * N / (time * 1e9),
      3.0 * N * sizeof(real_t) / (time * 1e9));
    status = hipblasDestroy(handle);
  }

  // =========================
  // (10) perform result comparison
  //      we need to re-run the CPU code because
  //      it has been executed 1000 times before
  // =========================
  int errorCount = 0;
  for (size_t i = 0; i < N; ++i)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (real_t)(N - i + 1);
  }
  saxpy_serial(N, alpha, h_x, h_y);
  for (size_t i = 0; i < N; ++i)
  {
    if (abs(h_y[i] - h_z[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");


  // =========================
  // (11) clean up, free memory
  // =========================
  free(h_x);
  free(h_y);
  free(h_z);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
